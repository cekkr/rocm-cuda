// RUN: %run_test hipify "%s" "%t" %hipify_args 1 --hip-kernel-execution-syntax %clang_args

#pragma once
// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
// CHECK: #include "hip/hip_fp16.h"
#include "hip/hip_fp16.h"
/**
 * Allocate GPU memory for `count` elements of type `T`.
 */
template<typename T>
static T *gpuMalloc(size_t count) {
    T *ret = nullptr;
    // CHECK: hipMalloc(&ret, count * sizeof(T));
    hipMalloc(&ret, count * sizeof(T));
    return ret;
}

template<typename T>
__global__ void add(int n, T *x, T *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  // CHECK: half2 tmp = __float2half2_rn(0.0f);
  half2 tmp = __float2half2_rn(0.0f);
  // CHECK: float max_val = fmax((float)reinterpret_cast<half&>(tmp.x), (float)reinterpret_cast<half&>(tmp.y));
  float max_val = fmax((float)reinterpret_cast<half&>(tmp.x), (float)reinterpret_cast<half&>(tmp.y));
  // CHECK: float min_val = fmin((float)reinterpret_cast<half&>(tmp.x), (float)reinterpret_cast<half&>(tmp.y));
  float min_val = fmin((float)reinterpret_cast<half&>(tmp.x), (float)reinterpret_cast<half&>(tmp.y));
  for (int i = index; i < n; i += stride)
      y[i] = max_val - min_val + x[i] + y[i];
}

int main(int argc, char* argv[]) {
    size_t numElements = 50;
    float *A = gpuMalloc<float>(numElements);
    float* B = gpuMalloc<float>(numElements);
    for (int i = 0; i < numElements; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }
    int blockSize = 512;
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    dim3 dimGrid(numBlocks, 1, 1);
    dim3 dimBlock(blockSize, 1, 1);
    // CHECK: hipLaunchKernelGGL(HIP_KERNEL_NAME(add<float>), dim3(dimGrid), dim3(dimBlock), 0, 0, numElements, A, B);
    add<float><<<dimGrid, dimBlock>>>(numElements, A, B);
    // CHECK: hipDeviceSynchronize();
    hipDeviceSynchronize();
    float maxError = 0.0f;
    for (int i = 0; i < numElements; ++i)
        maxError = fmax(maxError, fabs(B[i] - 3.0f));
    // CHECK: hipFree(A);
    hipFree(A);
    // CHECK: hipFree(B);
    hipFree(B);
    if (maxError == 0.0f)
        return 0;
    return -1;
}
